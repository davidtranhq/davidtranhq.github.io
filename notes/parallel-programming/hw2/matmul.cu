
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <cassert>
#include <ctime>

using namespace std;

struct cuda_exception {
    explicit cuda_exception(const char *err) : error_info(err) {}
    explicit cuda_exception(const string &err) : error_info(err) {}
    string what() const throw() { return error_info; }

    private:
    string error_info;
};

void checkCudaError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        string error_info(msg);
        error_info += " : ";
        error_info += hipGetErrorString(err);
        throw cuda_exception(error_info);
    }
}

template<typename T>
void random_matrix(T *M, size_t height, size_t width, int p = 2) {
    for(size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            M[i * width + j] = rand() % p; 
        }
    }
}

template<typename T>
void print_matrix(const T *M, size_t height, size_t width) {
    if (height >= 32 || width >= 32) {
        cout << "a matrix of height " << height << ", of width " << width << endl;
        return;
    }

    for(size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            cout << M[i * width + j] << "   ";
        }
        cout << endl;
    }
    cout << endl;
}

#define BLOCK_SIZE 16

/**
 * CUDA kernel for matrix multiplication, blockwise multiplcation
 *
 * @C, the output matrix C = A * B
 * @A, the first input matrix
 * @B, the second input matrix
 * @wa, is the width of A
 * @wb, is the width of B
 *
 * returns void.
 * 
 * The heights of A and B are not passed since wa, wb and
 * the  block indices are sufficient to dertmine the leading
 * position of the matrix tile that a given thread is working on 
 *
 * One thread block is computing a tile in C, thus, it
 * computes a dot-product of tile vectors.
 * Each element in C is computed by one thread of one
 * thread block.
 * Each thread computes its C-element as a sum of 
 * "small" dot-products, where each of these dot-products
 * is a dot-product of two vectors in a tile.
 * 
 */
    template <typename T>
__global__ void matrix_mul_ker(T* C, const T *A, const T *B, 
        size_t wa, size_t wb) 
{
    // Block index; WARNING: should be at most 2^16 - 1
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first submatrix of A processed by the block
    // This is the y-coordinate of the NW corner of the working tile
    int aBegin = wa * BLOCK_SIZE * by;

    // Index of the last submatrix of A processed by the block
    // This is the y-coordinate of the SE corner of the working tile
    int aEnd = aBegin + wa - 1;

    // Step size used to iterate through the submatrices of A
    // 
    int aStep = BLOCK_SIZE;

    // Index of the first submatrix of B processed by the block
    // This is the x-coordinate of the NW corner of the working tile
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the submatrices of B
    int bStep = BLOCK_SIZE * wb;

    // The element of the block submatrix that is computed by the thread
    // WARNING: This is a local variable for the working thread
    int Csub = 0;

    // Loop over all the submatrices of A and B required to
    // compute the block submatrix
    // This loop iterates  through the tiles in A ndn B that
    // contribute to the working tile of C
    for(int a = aBegin, b = bBegin;  a <= aEnd; a += aStep, b += bStep) {
        // shared memory for the submatrix of A
        __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];

        // shared memory for the submatrix of B
        __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from global memory to shared memory
        // each thread loads one element of each matrix
        As[ty][tx] = A[a + wa * ty + tx];
        Bs[ty][tx] = B[b + wb * ty + tx];

        // synchronize to make sure the matrices are loaded 
        __syncthreads();

        // Multiply the two matrices together
        // each thread computes one element of the block submatrix
        for(int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }
        // synchronize to make sure that the preceding computation is 
        // done before loading two new submatrices of A dnd B in the next iteration
        __syncthreads();		   
    }   
    // Write the block submatrix to global memory;
    // each thread writes one element
    int c = wb * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wb * ty + tx] = Csub;
}

template <typename T>
void matrix_mul_dev(T* C, const T* A, const T* B, int ha, int wa, int wb) {

    assert(wa % BLOCK_SIZE == 0);
    assert(wb % BLOCK_SIZE == 0);

    // load A and B to the device
    size_t mem_size = ha * wa * sizeof(T);

    T* Ad;
    hipMalloc((void **)&Ad, mem_size);
    checkCudaError("allocate GPU memory for the first matrix");
    hipMemcpy(Ad, A, mem_size, hipMemcpyHostToDevice);

    T* Bd;
    mem_size = wa * wb * sizeof(T);
    hipMalloc((void **)&Bd, mem_size);
    checkCudaError("allocate GPU memory for the second matrix");
    hipMemcpy(Bd, B, mem_size, hipMemcpyHostToDevice);

    // allocate C on the device
    T* Cd;
    mem_size = ha * wb * sizeof(int);
    hipMalloc((void**)&Cd, mem_size);
    checkCudaError("allocate GPU memory for the output matrix");

    // compute the execution configure
    // assume that the matrix dimensions are multiples of BLOCK_SIZE
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    size_t dgx = wb / dimBlock.x;
    size_t dgy = ha / dimBlock.y;
    dim3 dimGrid(dgx, dgy);

    // launch the device computation
    matrix_mul_ker<<<dimGrid, dimBlock>>>(Cd, Ad, Bd, wa, wb);
    hipDeviceSynchronize();
    checkCudaError("call the matrix multiplication kernel");

    // read C from the device 
    hipMemcpy(C, Cd, mem_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}

/**
 * Returns the time spent in seconds
 */
template <typename T>
double matrix_mul_gpu(T* C, const T* A, const T* B, int ha, int wa, int wb) {
    clock_t t1 = clock();

    // do the multiplication
    matrix_mul_dev(C, A, B, ha, wa, wb);

    clock_t t2 = clock();
    return (t2 - t1) / double(CLOCKS_PER_SEC);
}

/**
 * ha = 2^eha,
 * wa = 2^ewa,
 * wb = 2^ewb;
 *
 * If no parameter entered, then default values are used.
 * If one parameter rentered, it is eha = ewa = ewb = argv[1].
 * If three parameters rentered, there are eha, ewa, and ewb, respectively.
 *
 */
int matrix_mul_test(int argc, char **argv) {
    int *A, *B, *C;
    size_t eha = 4;
    size_t ewa = 4;
    size_t ewb = 4;

    if (argc == 2) {
        eha = ewa = ewb = atoi(argv[1]);
    } else if (argc >= 3) { 
        eha = atoi(argv[1]);
        ewa = atoi(argv[2]);
        ewb = atoi(argv[3]);
    }

    size_t ha = (1L << eha);
    size_t wa = (1L << ewa);
    size_t wb = (1L << ewb);

    try {
        A = new int[ha * wa];
        B = new int[wa * wb];
        C = new int[ha * wb];
        random_matrix(A, ha, wa);
        random_matrix(B, wa, wb);

        // timing gpu based method 
        cout << matrix_mul_gpu(C, A, B, ha, wa, wb) << "(seconds)" << endl;

    } catch (cuda_exception& err) {
        cout << err.what() << endl;
        delete [] A;
        delete [] B;
        delete [] C;
        return EXIT_FAILURE;
    } catch (...) {
        delete [] A;
        delete [] B;
        delete [] C;
        cout << "unknown exeception" << endl;
        return EXIT_FAILURE;
    }

    print_matrix(A, ha, wa);
    print_matrix(B, wa, wb);
    print_matrix(C, ha, wb);

    delete [] A;
    delete [] B;
    delete [] C;
    return 0;
}

int main(int argc, char** argv) {
    matrix_mul_test(argc, argv);
    return 0;
}

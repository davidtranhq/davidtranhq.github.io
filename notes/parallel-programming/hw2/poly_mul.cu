#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <string>
#include <chrono>

#include "poly_mul.hpp"

template <typename T>
void naive_polynomial_mul(
    T const *poly1,
    T const *poly2,
    T       *result, 
    int   const  degree
) {
    for (int i = 0; i <= 2 * degree; i++) {
        T sum = 0;
        for (int j = 0; j <= i; j++) {
            if (j <= degree && (i - j) <= degree) {
                sum += poly1[j] * poly2[i - j];
            }
        }
        result[i] = sum;
    }
}

void test_correctness() {

    auto generate_polynomial = [](int degree) {
        static auto rd = std::random_device {};
        static auto gen = std::mt19937(rd());
        static auto dis = std::uniform_int_distribution<int>(-1, 1);
        auto poly = std::vector<int>(degree + 1);
        for (int i = 0; i < degree + 1; ++i) {
            poly[i] = dis(gen);
        }
        return poly;
    };

    std::cout << "Running correctness tests...\n";
    std::vector<int> test_degrees = {0, 1, 1024, 2048, (1 << 16)};
    for (int const degree : test_degrees) {
        auto poly1 = generate_polynomial(degree);
        auto poly2 = generate_polynomial(degree);
        auto result = std::vector<int>(2 * degree + 1);
        auto expected = std::vector<int>(2 * degree + 1);

        naive_polynomial_mul(poly1.data(), poly2.data(), result.data(), degree);
        polynomial_mul(poly1.data(), poly2.data(), expected.data(), degree, 256);

        if (result != expected) {
            std::cerr << "[TEST FAILED] Test failed for degree " << degree
                << "\nPolynomial 1: ";
            for (auto const coeff : poly1) std::cerr << coeff << " ";
            std::cerr << "\nPolynomial 2: ";
            for (auto const coeff : poly2) std::cerr << coeff << " ";
            std::cerr << "\nExpected: ";
            for (auto const coeff : expected) std::cerr << coeff << " ";
            std::cerr << "\nGot: ";
            for (auto const coeff : result) std::cerr << coeff << " ";
            std::cerr << '\n';
            return;
        }

        std::cout << "[TEST PASSED] Test passed for degree " << degree << "\n";
    }
    std::cout << "\nPASSED ALL TESTS\n";
}

void test_naive_performance(int degree, int block_size) {
    const int numCoefficients = 2 * degree + 1;

    // Define polynomials
    int poly1[numCoefficients];
    int poly2[numCoefficients];
    std::fill(poly1, poly1 + numCoefficients, 1.0f);
    std::fill(poly2, poly2 + numCoefficients, 1.0f);
    int result[numCoefficients];

    // use std::chrono to measure time it takes to run naive_polynomial_mul
    auto start = std::chrono::high_resolution_clock::now();
    naive_polynomial_mul(poly1, poly2, result, degree);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    // print degree, block size, and elapsed time
    std::cout << "(Naive) Degree: " << degree << ", Block Size: " << block_size << ", Elapsed Time: " << elapsed.count() << " s" << std::endl;

}

void test_performance(int degree, int block_size, PolynomialMulMethod method) {
    const int numCoefficients = 2 * degree + 1;

    // Define polynomials
    int poly1[numCoefficients];
    int poly2[numCoefficients];
    std::fill(poly1, poly1 + numCoefficients, 1.0f);
    std::fill(poly2, poly2 + numCoefficients, 1.0f);
    int result[numCoefficients];

    // Create CUDA events for profiling
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Perform polynomial multiplication using the given method
    hipEventRecord(start); // Record start time
    polynomial_mul(poly1, poly2, result, degree, block_size, method);
    hipEventRecord(stop); // Record stop time

    // Synchronize to make sure all CUDA operations are completed
    hipDeviceSynchronize();

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print result
    std::string method_name;
    switch (method) {
        case PolynomialMulMethod::naive:
            method_name = "naive";
            break;
        case PolynomialMulMethod::fast:
            method_name = "fast";
            break;
    }

    // print method name, degree, block size, and elapsed time
    std::cout << "Method: " << method_name << ", Degree: " << degree << ", Block Size: " << block_size << ", Elapsed Time: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);

}


int main() {
    test_correctness();

    // test_naive_performance(3, 256);
    // test_performance(3, 256);
    return 0;
}
